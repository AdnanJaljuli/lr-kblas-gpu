/**
 * @copyright (c) 2012- King Abdullah University of Science and
 *                      Technology (KAUST). All rights reserved.
 **/


/**
 * @file src/batch_triangular/Xgemm_batch.cu

 * KBLAS is a high performance CUDA library for subset of BLAS
 *    and LAPACK routines optimized for NVIDIA GPUs.
 * KBLAS is provided by KAUST.
 *
 * @version 2.0.0
 * @author Ali Charara
 * @date 2017-11-13
 **/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#include "kblas.h"
#include "operators.h"
#include <typeinfo>

// #ifdef USE_MAGMA
// #include "magma.h"
// #endif

#include "kblas_struct.h"
#include "kblas_prec_def.h"

#include "kblas_common.h"
#include "batch_common.ch"
#include "Xhelper_funcs.ch"
#include "Xgemm_batch_core.cuh"

//=================================================================================
//Non-Strided form

/**
 * Uniform-size batch non-strided GEMM wrapper
 *
 * Workspace needed: device pointers
 * Use gemm_batch_offset_wsquery_core()
 * A, B, C: host pointer to array of device pointers to device buffers
 */
int kblas_gemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      const int m, const int n, const int k,
                      const TYPE alpha,
                      const TYPE** A, int A_row_off, int A_col_off, int lda,
                      const TYPE** B, int B_row_off, int B_col_off, int ldb,
                      const TYPE beta,
                            TYPE** C, int C_row_off, int C_col_off, int ldc,
                      int batchCount){
  return Xgemm_batch_core(handle,
                          transA, transB,
                          m, n, k,
                          alpha,
                          A, A_row_off, A_col_off, lda,
                          B, B_row_off, B_col_off, ldb,
                          beta,
                          C, C_row_off, C_col_off, ldc,
                          batchCount);
}

/**
 *  Uniform-size batch non-strided GEMM wrapper
 *
 *  Workspace needed: none
 *  @param A, B, C: host pointer to array of device pointers to device buffers
 */
int kblas_gemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      const int m, const int n, const int k,
                      const TYPE alpha,
                      const TYPE** A, int lda,
                      const TYPE** B, int ldb,
                      const TYPE beta,
                            TYPE** C, int ldc,
                      int batchCount){
  return Xgemm_batch_core(handle,
                          transA, transB,
                          m, n, k,
                          alpha,
                          A, lda,
                          B, ldb,
                          beta,
                          C, ldc,
                          batchCount);
}
extern "C"
int kblasXgemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      const int m, const int n, const int k,
                      const TYPE alpha,
                      const TYPE** A, int lda,
                      const TYPE** B, int ldb,
                      const TYPE beta,
                            TYPE** C, int ldc,
                      int batchCount){
  return Xgemm_batch_core(handle,
                          transA, transB,
                          m, n, k,
                          alpha,
                          A, lda,
                          B, ldb,
                          beta,
                          C, ldc,
                          batchCount);
}

//==============================================================================================
//Strided form

//TODO IMPORTANT: stride should be long long int since it is a memory address measure


/**
 * Uniform-size batch strided GEMM wrapper
 *
 * Workspace needed= ( __CUDACC_VER_MAJOR__ < 8 ) ? device pointers : none
 * @param A, B, C: host pointers to device buffers
 */
int kblas_gemm_batch( kblasHandle_t handle,
                      char transA, char transB,
                      const int m, const int n, const int k,
                      const TYPE alpha,
                      const TYPE* A, int lda, long strideA,
                      const TYPE* B, int ldb, long strideB,
                      const TYPE beta,
                            TYPE* C, int ldc, long strideC,
                      int batchCount){
  return Xgemm_batch_core(handle,
                          transA, transB,
                          m, n, k,
                          alpha,
                          A, lda, strideA,
                          B, ldb, strideB,
                          beta,
                          C, ldc, strideC,
                          batchCount);
}

// A, B, C: host pointers to device buffers
extern "C"
int kblasXgemm_batch_strided( kblasHandle_t handle,
                              char transA, char transB,
                              const int m, const int n, const int k,
                              const TYPE alpha,
                              const TYPE* A, int lda, long strideA,
                              const TYPE* B, int ldb, long strideB,
                              const TYPE beta,
                                    TYPE* C, int ldc, long strideC,
                              int batchCount){
  return Xgemm_batch_core(handle,
                          transA, transB,
                          m, n, k,
                          alpha,
                          A, lda, strideA,
                          B, ldb, strideB,
                          beta,
                          C, ldc, strideC,
                          batchCount);
}
