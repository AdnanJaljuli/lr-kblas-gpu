/**
  --* (C) Copyright 2013 King Abdullah University of Science and Technology
  Authors:
  Ali Charara (ali.charara@kaust.edu.sa)
  David Keyes (david.keyes@kaust.edu.sa)
  Hatem Ltaief (hatem.ltaief@kaust.edu.sa)

  Redistribution  and  use  in  source and binary forms, with or without
  modification,  are  permitted  provided  that the following conditions
  are met:

  * Redistributions  of  source  code  must  retain  the above copyright
  * notice,  this  list  of  conditions  and  the  following  disclaimer.
  * Redistributions  in  binary  form must reproduce the above copyright
  * notice,  this list of conditions and the following disclaimer in the
  * documentation  and/or other materials provided with the distribution.
  * Neither  the  name of the King Abdullah University of Science and
  * Technology nor the names of its contributors may be used to endorse
  * or promote products derived from this software without specific prior
  * written permission.
  *
  THIS  SOFTWARE  IS  PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
  ``AS IS''  AND  ANY  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
  LIMITED  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A  PARTICULAR  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL,  EXEMPLARY,  OR  CONSEQUENTIAL  DAMAGES  (INCLUDING,  BUT NOT
  LIMITED  TO,  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA,  OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY  OF  LIABILITY,  WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF  THIS  SOFTWARE,  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
**/
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#include <typeinfo>

#include "kblas.h"
#include "kblas_struct.h"
#include "operators.h"
#include "defs.h"
#include "kblas_common.h"
#include "batch_common.ch"

//==============================================================================================
#include "Xblas_core.ch"
#include "Xhelper_funcs.ch"
#include "Xtrmm_batch_drivers.cuh"

//==============================================================================================
//Non-Strided form

// workspace needed: device pointers
// A, B: host pointer to array of device pointers to device buffers
int kblas_trmm_batch(kblasHandle_t handle,
                     char side, char uplo, char trans, char diag,
                     const int m, const int n,
                     const TYPE alpha,
                     const TYPE** A, int lda,
                           TYPE** B, int ldb,
                    int batchCount){

  KBlasWorkspaceState ws_needed;
  trmm_batch_wsquery_core<false>( batchCount,
                                  side, m, n,
                                  (kblasWorkspaceState_t)&ws_needed);

  bool suffWorkspace = (ws_needed.d_ptrs_bytes <= handle->work_space.allocated_ws_state.d_ptrs_bytes);

  if(!suffWorkspace){
    return KBLAS_InsufficientWorkspace;
  }

  return Xtrmm_batch_core<TYPE, TYPE**, false>(
                          handle,
                          side, uplo, trans, diag,
                          m, n,
                          alpha,
                          (TYPE**)A, 0, 0, lda, (long)0,
                          (TYPE**)B, 0, 0, ldb, (long)0,
                          batchCount);
}

extern "C" {

// workspace needed: device pointers
// A, B: host pointer to array of device pointers to device buffers
int kblasXtrmm_batch(kblasHandle_t handle,
                     char side, char uplo, char trans, char diag,
                     const int m, const int n,
                     const TYPE alpha,
                     const TYPE** A, int lda,
                           TYPE** B, int ldb,
                    int batchCount){

  KBlasWorkspaceState ws_needed;
  trmm_batch_wsquery_core<false>( batchCount,
                                  side, m, n,
                                  (kblasWorkspaceState_t)&ws_needed);

  bool suffWorkspace = (ws_needed.d_ptrs_bytes <= handle->work_space.allocated_ws_state.d_ptrs_bytes);

  if(!suffWorkspace){
    return KBLAS_InsufficientWorkspace;
  }

  return Xtrmm_batch_core<TYPE, TYPE**, false> (
                          handle,
                          side, uplo, trans, diag,
                          m, n,
                          alpha,
                          (TYPE**)A, 0, 0, lda, (long)0,
                          (TYPE**)B, 0, 0, ldb, (long)0,
                          batchCount);
}

} //extern "C"

//==============================================================================================
//Strided form
// template<>

// workspace needed: device pointers
// A, B: host pointer to array of device pointers to device buffers
int kblas_trmm_batch(kblasHandle_t handle,
                     char side, char uplo, char trans, char diag,
                     const int m, const int n,
                     const TYPE alpha,
                     const TYPE* A, int lda, long strideA,
                           TYPE* B, int ldb, long strideB,
                    int batchCount){

  KBlasWorkspaceState ws_needed;
  trmm_batch_wsquery_core<true>(batchCount,
                                side, m, n,
                                (kblasWorkspaceState_t)&ws_needed);

  bool suffWorkspace = (ws_needed.d_ptrs_bytes <= handle->work_space.allocated_ws_state.d_ptrs_bytes);

  if(!suffWorkspace){
    return KBLAS_InsufficientWorkspace;
  }

  return Xtrmm_batch_core<TYPE, TYPE*, true>(
                          handle,
                          side, uplo, trans, diag,
                          m, n,
                          alpha,
                          (TYPE*)A, 0, 0, lda, strideA,
                          (TYPE*)B, 0, 0, ldb, strideB,
                          batchCount);
}

extern "C" {

// workspace needed: device pointers
// A, B: host pointer to device buffers
int kblasXtrmm_batch_strided(kblasHandle_t handle,
                             char side, char uplo, char trans, char diag,
                             const int m, const int n,
                             const TYPE alpha,
                             const TYPE* A, int lda, long strideA,
                                   TYPE* B, int ldb, long strideB,
                             int batchCount){

  KBlasWorkspaceState ws_needed;
  trmm_batch_wsquery_core<true>(batchCount,
                                side, m, n,
                                (kblasWorkspaceState_t)&ws_needed);

  bool suffWorkspace = (ws_needed.d_ptrs_bytes <= handle->work_space.allocated_ws_state.d_ptrs_bytes);

  if(!suffWorkspace){
    return KBLAS_InsufficientWorkspace;
  }

  return Xtrmm_batch_core<TYPE, TYPE*, true> (
                          handle,
                          side, uplo, trans, diag,
                          m, n,
                          alpha,
                          (TYPE*)A, 0, 0, lda, strideA,
                          (TYPE*)B, 0, 0, ldb, strideB,
                          batchCount);
}

}//extern C
